#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"

#define NUM_THREADS 1000000
#define ARRAY_SIZE 100 

#define BLOCK_WIDTH 1000 

void print_array(int *array, int size){
    printf("{ ");
    for (int i = 0;i<size;i++){
        printf("%d ", array[i]);
    }
    printf("}\n");
}

__global__ void increment_naive(int *g){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    i = i % ARRAY_SIZE;
    g[i] = g[i] + 1;
}


__global__ void increment_atomic(int *g){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    i = i % ARRAY_SIZE;
    atomicAdd(&g[i],1);
}


int main(int argc, char** argv){
    
    GpuTimer timer;
    printf("%d total threads in %d blocks writing into %d array elements\n",
          NUM_THREADS,NUM_THREADS / BLOCK_WIDTH, ARRAY_SIZE);

    int h_array[ARRAY_SIZE];
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    int* d_array;

    hipMalloc((void**) &d_array, ARRAY_BYTES);
    hipMemset((void*) d_array, 0, ARRAY_BYTES);

    timer.Start();

    increment_atomic<<<NUM_THREADS/BLOCK_WIDTH,BLOCK_WIDTH>>>(d_array);

    timer.Stop();

    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);

    print_array(h_array,ARRAY_SIZE);

    printf("Time elapsed = %g ms\n",timer.Elapsed());

    hipFree(d_array);
    return 0;

    
}
